
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void addition(int *a, int*b, int n)
{
  int tid=threadIdx.x;
  int sum=0;
  for(int i=0;i<n;i++)
  {
      sum+=a[i];
  } 
  b[tid]=sum; 
}

int main()
{
  int n=1000;
  int *a=(int*)malloc(n*sizeof(int));
  hipEvent_t start, end;
  for(int i=0;i<n;i++)
  {
      a[i]=i+1;
  }
  hipEventCreate(&start);
  hipEventCreate(&end);
  int *dev_a,  *dev_b;
  int size=n*sizeof(int);
  hipMalloc(&dev_a,size);
  hipMalloc(&dev_b,sizeof(int));
  hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
  hipEventRecord(start);
  addition<<<1, n>>>(dev_a, dev_b, n);
  int *sum=(int *)malloc(sizeof(int));
  hipEventRecord(end);
  float time=0;
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);
  hipMemcpy(sum, dev_b, sizeof(int),hipMemcpyDeviceToHost);
  cout<<"\nAddition is : "<<sum[0];
  cout<<"\nTime taken : "<<time;
  return 0;
}
