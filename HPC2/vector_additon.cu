
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void addition(int *a,int *b,int *c,int n)
{
	int large_id=blockIdx.x*blockDim.x+threadIdx.x;

	while(large_id<n)
	{
		c[large_id]=a[large_id]+b[large_id];
		large_id+=blockDim.x*gridDim.x;
	}

}

int main()
{
	int *a,*b,*c;

	int n=20;

	a=(int*)malloc(n*sizeof(int));
	b=(int*)malloc(n*sizeof(int));
	c=(int*)malloc(n*sizeof(int));

	for(int i=0;i<n;i++)
	{
		a[i]=i+1;
		b[i]=i+1;
		c[i]=0;
	}

	hipEvent_t start,end;
	int size=n*sizeof(int);

	int *dev_a,*dev_b,*dev_c;

	hipMalloc(&dev_a,size);
	hipMalloc(&dev_b,size);
	hipMalloc(&dev_c,size);

	hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,size,hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);

	addition<<<128,128>>>(dev_a,dev_b,dev_c,n);

	hipEventRecord(end);
	hipEventSynchronize(end);

	float time=0;
	hipEventElapsedTime(&time,start,end);

	hipMemcpy(c,dev_c,size,hipMemcpyDeviceToHost);
	for(int i = 0; i < n; i++) {
		cout<<a[i]<<"+"<<b[i]<<"="<<c[i]<<endl;
	}
	cout<<"\n Time elapsed:"<<time<<endl;

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);



}