#include "hip/hip_runtime.h"
#include<iostream>
using namespace std;

__global__ void vectMat(int*a,int*b,int *c,int n)
{
	int row=blockIdx.y*blockDim.y+threadIdx.y;

	int sum=0;
	for(int j=0;j<n;j++)
	{
		sum+=a[row*n+j]*b[j];
	}
	c[row]=sum;

}
int main()
{
	int n=3;

	int*a,*b,*c;
	a=new int[n*n];
	b=new int[n];
	c=new int[n];
	
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			a[i*n+j]=i+j;
		}
	}
	
	for(int i=0;i<n;i++)
	{
		b[i]=i+1;
	}
	
	cout<<"Matrix A is: "<<endl;
	for(int i = 0; i < n; i++) {
		for(int j = 0; j < n; j++) {
			cout << "a[" << i * n + j << "] = " << a[i * n + j] << " ";
		}
		cout << endl;
	}

	cout<<"Vector B is: "<<endl;
	for(int i = 0; i < n; i++) {
		cout << "b[" << i << "] = " <<b[i] << " ";
	}
	cout<<endl;

	int size=n*sizeof(int);

	hipEvent_t start,end;

	int* dev_a,*dev_b,*dev_c;
	hipMalloc(&dev_a,n*size);
	hipMalloc(&dev_b,size);
	hipMalloc(&dev_c,size);

	hipEventCreate(&start);
	hipEventCreate(&end);

	hipMemcpy(dev_a,a,n*size,cudaMemcpyHostTODevice);
	hipMemcpy(dev_b,b,size,hipMemcpyHostToDevice);
	

	dim3 grid_dim(n,n,1);
	hipEventRecord(start);

	vectMat<<<grid_dim,1>>>(dev_a,dev_b,dev_c,n);
	hipEventRecord(end);
	hipEventSynchronize(end);

	hipMemcpy(dev_c,c,size,hipMemcpyDeviceToHost);

	float time=0;
	hipEventElapsedTime(&time,start,end);
	cout << "Output: " << endl;
	for(int i = 0; i < n; i++) {
		cout<< "c[" << i << "] = " << c[i] <<" ";
	}

	cout<<"\n Time elpased"<<time<<endl;



}